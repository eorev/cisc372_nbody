#include "hip/hip_runtime.h"
#include "config.h"
#include "vector.h"
#include <hip/hip_runtime.h>
#include <math.h>

#define NUMELEMENTS 1024
#define BLOCK_SIZE 16

// Function to create a vector3
__device__ vector3 make_vector3(double x, double y, double z) {
  vector3 v;
  v.x = x;
  v.y = y;
  v.z = z;
  return v;
}

// Function to compute dot product
__device__ double dot_product(vector3 a, vector3 b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}

// Function to multiply vector by a scalar
__device__ vector3 scalar_mult(vector3 v, double scalar) {
  return make_vector3(v.x * scalar, v.y * scalar, v.z * scalar);
}

// Function to add two vectors
__device__ vector3 vector_add(vector3 a, vector3 b) {
  return make_vector3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void computeAccelerationMatrix(vector3 *accels, vector3 *d_hPos,
                                          double *d_mass) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < NUMELEMENTS && j < NUMELEMENTS) {
    vector3 distance;
    double magnitude_sq, magnitude, accelmag;

    if (i != j) {
      for (int k = 0; k < 3; k++) {
        distance[k] = d_hPos[j][k] - d_hPos[i][k];
      }
      magnitude_sq = distance[0] * distance[0] + distance[1] * distance[1] +
                     distance[2] * distance[2];
      magnitude = sqrt(magnitude_sq);
      accelmag = GRAV_CONSTANT * d_mass[j] / magnitude_sq;

      for (int k = 0; k < 3; k++) {
        double accelComponent = accelmag * distance[k] / magnitude;
        accels[i * NUMELEMENTS + j][k] = accelComponent;
        accels[j * NUMELEMENTS + i][k] = -accelComponent;
      }
    } else {
      for (int k = 0; k < 3; k++) {
        accels[i * NUMELEMENTS + j][k] = 0;
      }
    }
  }
}

__global__ void updateVelocityPosition(vector3 *accels, vector3 *d_hPos,
                                       vector3 *d_hVel) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < NUMELEMENTS) {
    vector3 totalAccel = {0, 0, 0};

    for (int j = 0; j < NUMELEMENTS; j++) {
      for (int k = 0; k < 3; k++) {
        totalAccel[k] += accels[i * NUMELEMENTS + j][k];
      }
    }

    for (int k = 0; k < 3; k++) {
      d_hVel[i][k] += totalAccel[k] * INTERVAL;
      d_hPos[i][k] += d_hVel[i][k] * INTERVAL;
    }
  }
}

void compute(vector3 *d_hPos, vector3 *d_hVel, double *d_mass) {
  vector3 *d_accels;
  hipMalloc((void **)&d_accels, sizeof(vector3) * NUMELEMENTS * NUMELEMENTS);

  dim3 dimBlock(16, 16);
  dim3 dimGrid((NUMELEMENTS + dimBlock.x - 1) / dimBlock.x,
               (NUMELEMENTS + dimBlock.y - 1) / dimBlock.y);

  computeAccelerationMatrix<<<dimGrid, dimBlock>>>(d_accels, d_hPos, d_mass);
  updateVelocityPosition<<<(NUMELEMENTS + 255) / 256, 256>>>(d_accels, d_hPos,
                                                             d_hVel);

  hipFree(d_accels);
}